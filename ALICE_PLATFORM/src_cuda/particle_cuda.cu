#include "hip/hip_runtime.h"
//#define _VERBOSE ;

#ifndef MIN
#define MIN(x,y) (((x) < (y)) ? (x) : (y))
#endif

#ifndef MAX
#define MAX(x,y) (((x) > (y)) ? (x) : (y))
#endif

#define transform \
dev_pos[idx] =  make_float4(p[idx].p.x,p[idx].p.y,p[idx].p.z,1.0);\
dev_pos[idx].w =  d;\
if(rotate)dev_pos[idx] = ArbitraryRotate2(p[idx].p,angle , t , make_float3(t.x,t.y,t.z + 1)) ;\
dev_pos[idx].w =  d;\
if(translate)\
{\
dev_pos[idx].x -= t.x -128.0 * i ;\
dev_pos[idx].y -= t.y ;\
dev_pos[idx].z = 0.0f ;\
dev_pos[idx].w =  d;\
}\


//dev_pos[idx].z =  10;\
//dev_pos[idx].w =  p[idx].p.z;\

//dev_pos[idx].z = 10.0f ;\
//#include "host_config.h"
//#include "builtin_types.h"
#include "hip/channel_descriptor.h"
#include "hip/hip_runtime_api.h"
#include "driver_functions.h"
//#include "host_defines.h"
//#include "vector_functions.h"
//#include "hip/hip_vector_types.h"
#include <hip/hip_runtime.h>




#include <stdio.h>
#include <iostream>

#ifndef _PARTICLE_CUDA
#define _PARTICLE_CUDA

typedef struct {
	double m;   /* 8 Mass                          */
	float3 p;      /*  12 Position                      */
	float3 v;      /* 12 Velocity                      */
	float3 f;      /* 12 Force                         */
	int fixed;  /* 4 Fixed point or free to move   */
	//
	double dummy1[2]; //16
	//double dummy2; //8 

} PARTICLE;

typedef struct {
	float3 dpdt; // 12
	float3 dvdt; //12 

	double dummy1; // 8
} PARTICLEDERIVATIVES;

typedef struct {
	double gravitational; //8
	double viscousdrag; //8 
} PARTICLEPHYS;

typedef struct {
	int from; //4 
	int to; //4 
	double springconstant; //8
	float3 springForce ; // 12
	double dampingconstant; //8 
	double restlength; //8

	float dummy[4] ; //20 

} PARTICLESPRING;

typedef struct
{
	float x,y,z ;
}point;

typedef struct
{
	point min ;
	point max ;
}bbox;




#endif

// ---------------

int nparticles = 0;
PARTICLE *particles;
PARTICLE *particles_buf;
int nsprings = 0;
PARTICLESPRING *springs;
PARTICLEPHYS physical;
PARTICLEDERIVATIVES *deriv;

int *ParticleCentricSpringIds ; 
int4 *ParticleCentricSpringIds_baseAddress ; 
int baseAddress = 0 ;

int *fixed0 ;
int *fixed1 ;
int *fixed2 ;
int *fixed3 ;
int n_f0,n_f1,n_f2,n_f3 ;

float3 lamp0,lamp1,lamp2,lamp3 ;

int numBlocks = 0 ;

//bool cudaInited = false ;
// ------------ 

float3 *d_vbo_pos;

bool dMat_init = false ;
bool dPos_init = false ;
bool d_vbo_init = false ;


int d_iterations = 0 ;;

#define timerStart \
hipEventCreate(&start); \
hipEventCreate(&stop); \
hipEventRecord( start, 0 ); \

#define timerEnd \
hipEventRecord( stop, 0 ); \
hipEventSynchronize( stop ); \
hipEventElapsedTime( &time, start, stop ); \

inline int cu_ConvertSMVer2Cores(int major, int minor)
{
	// Defines for GPU Architecture types (using the SM version to determine the # of cores per SM
	typedef struct {
		int SM; // 0xMm (hexidecimal notation), M = SM Major version, and m = SM minor version
		int Cores;
	} sSMtoCores;

	sSMtoCores nGpuArchCoresPerSM[] = 
	{ { 0x10,  8 }, // Tesla Generation (SM 1.0) G80 class
	{ 0x11,  8 }, // Tesla Generation (SM 1.1) G8x class
	{ 0x12,  8 }, // Tesla Generation (SM 1.2) G9x class
	{ 0x13,  8 }, // Tesla Generation (SM 1.3) GT200 class
	{ 0x20, 32 }, // Fermi Generation (SM 2.0) GF100 class
	{ 0x21, 48 }, // Fermi Generation (SM 2.1) GF10x class
	{   -1, -1 }
	};

	int index = 0;
	while (nGpuArchCoresPerSM[index].SM != -1) {
		if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor) ) {
			return nGpuArchCoresPerSM[index].Cores;
		}	
		index++;
	}
	printf("MapSMtoCores undefined SM %d.%d is undefined (please update to the latest SDK)!\n", major, minor);
	return -1;
}
int cu_gpuGetMaxGflopsDeviceId()
{
	int current_device     = 0, sm_per_multiproc  = 0;
	int max_compute_perf   = 0, max_perf_device   = 0;
	int device_count       = 0, best_SM_arch      = 0;
	hipDeviceProp_t deviceProp;
	hipGetDeviceCount( &device_count );

	// Find the best major SM Architecture GPU device
	while (current_device < device_count)
	{
		hipGetDeviceProperties( &deviceProp, current_device );
		if (deviceProp.major > 0 && deviceProp.major < 9999)
		{
			best_SM_arch = MAX(best_SM_arch, deviceProp.major);
		}
		current_device++;
	}

	// Find the best CUDA capable GPU device
	current_device = 0;
	while( current_device < device_count )
	{
		hipGetDeviceProperties( &deviceProp, current_device );
		if (deviceProp.major == 9999 && deviceProp.minor == 9999)
		{
			sm_per_multiproc = 1;
		}
		else
		{
			sm_per_multiproc = cu_ConvertSMVer2Cores(deviceProp.major, deviceProp.minor);
		}

		int compute_perf  = deviceProp.multiProcessorCount * sm_per_multiproc * deviceProp.clockRate;

		if( compute_perf  > max_compute_perf )
		{
			// If we find GPU with SM major > 2, search only these
			if ( best_SM_arch > 2 )
			{
				// If our device==dest_SM_arch, choose this, or else pass
				if (deviceProp.major == best_SM_arch)
				{
					max_compute_perf  = compute_perf;
					max_perf_device   = current_device;
				}
			}
			else
			{
				max_compute_perf  = compute_perf;
				max_perf_device   = current_device;
			}
		}
		++current_device;
	}
	return max_perf_device;
}

static void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if( hipSuccess != err) {
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
		//exit(EXIT_FAILURE); 
	}
} 





//////////////////////////////////////////////////////////////////////////


__device__ float Clamp(float value, float min, float max) 
{
	return value < min ? min : value > max ? max : value;
}

__device__ float Map(float value, float inputMin, float inputMax, float outputMin, float outputMax) 
{
	return ((value - inputMin) / (inputMax - inputMin) * (outputMax - outputMin) + outputMin);
}

__device__ float func(float x, float y)
{
	return ( cos(x) -sin(y)  ) ; //y - (1/(.5-pow(x,2))) ;//sin(x) - sin(pow(x,2));//tan(x)-tan(pow(x,2));//cos(x)-sin(y) ;// sin(x-y);//4*x - pow(x,2);
}

__device__ float dot(float3 a, float3 b) 
{
	return ( (a.x * b.x) + (a.y * b.y) + (a.z * b.z) );
}
__device__ float3 normalise(float3 a) 
{
	float inv_mag = rsqrt(dot(a,a));
	return make_float3( (a.x * inv_mag) , (a.y * inv_mag) , (a.z * inv_mag) );
}
__device__ float4 ArbitraryRotate2(float3 p,double theta,point p1,float3 p2)
{
	float4 q = make_float4(0.0,0.0,0.0,1.0);
	double costheta,sintheta;
	float3 r;

	r.x = p2.x - p1.x;
	r.y = p2.y - p1.y;
	r.z = p2.z - p1.z;
	p.x -= p1.x;
	p.y -= p1.y;
	p.z -= p1.z;
	r = normalise(r);

	costheta = cos(theta);
	sintheta = sin(theta);

	q.x += (costheta + (1 - costheta) * r.x * r.x) * p.x;
	q.x += ((1 - costheta) * r.x * r.y - r.z * sintheta) * p.y;
	q.x += ((1 - costheta) * r.x * r.z + r.y * sintheta) * p.z;

	q.y += ((1 - costheta) * r.x * r.y + r.z * sintheta) * p.x;
	q.y += (costheta + (1 - costheta) * r.y * r.y) * p.y;
	q.y += ((1 - costheta) * r.y * r.z - r.x * sintheta) * p.z;

	q.z += ((1 - costheta) * r.x * r.z - r.y * sintheta) * p.x;
	q.z += ((1 - costheta) * r.y * r.z + r.x * sintheta) * p.y;
	q.z += (costheta + (1 - costheta) * r.z * r.z) * p.z;

	q.x += p1.x;
	q.y += p1.y;
	q.z += p1.z;
	return(q);
}



__device__ bool isInside( float3 pt , bbox bx )
{
	return ( pt.x > bx.min.x && pt.x < bx.max.x &&
			pt.y > bx.min.y && pt.y < bx.max.y  ) ;
}

__device__ float sqDist( float3 pt , bbox bx )
{
	float2 mid = make_float2((bx.max.x+bx.min.x)*0.5 , (bx.max.y+bx.min.y)*0.5 );
	float2 t = make_float2(pt.x-mid.x ,pt.y-mid.y );

	float d1 = fabs(t.x*t.x + t.y*t.y) ;
	
	return d1;
}

__global__ void kernel_moveToNearest( PARTICLE *verts , PARTICLE *verts_buf , int nv , float disp )
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;


	if( idx < nv )
	{
		float3 p1,p2 , nearestP ;
		int nearestId ;
		float minD2 ;
		minD2 = pow(10.0f,10.0f) ;

		p1 = verts[idx].p ;

		if( !verts[idx].fixed )
		{
			for( int j =0 ; j < nv ; j++ )
			{
				if( idx == j )continue ;

				p2 = verts[j].p ;


				float3 v = make_float3( p2.x - p1.x ,p2.y - p1.y ,p2.z - p1.z ) ;
				float d2 = ( v.x*v.x + v.y*v.y + v.z*v.z) ;/*+ 1e-16*/; // calc distance squared to emitter - make sure it's not 0

				//if( d2 > disp * 10.0 )continue;

				if( d2 < minD2 && (p2.z < p1.z) && int(verts_buf[j].dummy1[0]) != (idx) )
				{
					minD2 = d2 ; 
					nearestP = p2 ;
					nearestId = j ;
				}

			}

			float3 v = make_float3( nearestP.x - p1.x ,nearestP.y - p1.y ,nearestP.z - p1.z ) ;
			float rsq = rsqrt(dot(v,v));   
			v.x *= rsq ;
			v.y *= rsq;
			v.z *= rsq ;// normalize v
			verts_buf[idx].p =  make_float3( p1.x + disp * v.x ,p1.y + disp * v.y ,p1.z + disp * v.z ) ;
			verts_buf[idx].dummy1[0] = float( nearestId ); ;
		}

		//rand[idx] = verts[idx].p.z ;
		//printf(" pts : %1.2f ,%1.2f ,%1.2f  \n" , verts_buf[idx].p.x ,verts_buf[idx].p.y ,verts_buf[idx].p.z ) ;
		//rand[idx] = verts[idx].v_acc ;

	}
}

__global__ void kernel_clearNeighborTags( PARTICLE *verts , int nv )
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;


	if( idx < nv )
	{
		verts[idx].dummy1[0] = 0 ;
	}
}

__global__ void kernel_calculateForces_springs(PARTICLE *p , int np,
									PARTICLEPHYS phys, PARTICLESPRING *s,int ns )
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;


	if( idx < ns ) 
	{
		/*s[idx].springForce.x = 0;
		s[idx].springForce.y = 0;
		s[idx].springForce.z = 0;*/
		int p1,p2;
		float3 down = make_float3(0.0,0.0,-1.0);
		float3 zero =  make_float3(0.0,0.0,0.0);
		float3 f;
		double len,dx,dy,dz;
		PARTICLESPRING sp = s[idx];
		
		float SC = sp.springconstant ;
		float RL = sp.restlength ;
		float DC = sp.dampingconstant ;

		p1 = sp.from;
		p2 = sp.to;

		

		PARTICLE pr1 =  p[p1] ;
		PARTICLE  pr2 =  p[p2] ;
		dx =  pr1.p.x -  pr2.p.x;
		dy =  pr1.p.y -  pr2.p.y;
		dz =  pr1.p.z -  pr2.p.z;
		len = /*rsqrt(dx*dx + dy*dy + dz*dz) * (dx*dx + dy*dy + dz*dz)  ;*/ sqrt(dx*dx + dy*dy + dz*dz);
		f.x  = SC  * (len - RL);
		f.x += DC * ( pr1.v.x -  pr2.v.x) * dx / len;
		f.x *= - dx / len;
		f.y  = SC  * (len - RL);
		f.y += DC * ( pr1.v.y -  pr2.v.y) * dy / len;
		f.y *= - dy / len;
		f.z  = SC  * (len - RL);
		f.z += DC * ( pr1.v.z -  pr2.v.z) * dz / len;
		f.z *= - dz / len;
		
		s[idx].springForce.x = f.x ;
		s[idx].springForce.y = f.y ;
		s[idx].springForce.z = f.z ;
		
		//if (!pr1.fixed) 
		//{
		//	p[p1].f.x += f.x;
		//	p[p1].f.y += f.y;
		//	p[p1].f.z += f.z;
		//	//p[p1] = pr1;
		//}
		//
		//if (!pr2.fixed)
		//{
		//	p[p2].f.x -= f.x;
		//	p[p2].f.y -= f.y;
		//	p[p2].f.z -= f.z;

		//	//p[p2] = pr2;
		//}


	}
}



__global__ void kernel_gatherPerParticleForceAndUpdate(PARTICLE *p,int np ,PARTICLESPRING *s ,PARTICLEPHYS phys ,int *PCSpringIds , int4 *PCSpringIds_baseAddress , double dt )
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	/*for( int idx =0 ; idx < np ; idx++ )*/
	if( idx < np )
	{
		if( !p[idx].fixed )
		{
			int4 baseID = PCSpringIds_baseAddress[idx];
			int baseAddress = baseID.x ;
			int fromCnt = baseID.y ;
			int toCnt = baseID.z ;
			/*printf(" per particle spring ids for vert ID %i \n : fromCnt %i , toCnt %i \n" , idx ,fromCnt , toCnt  ) ;*/
			float3 v = p[idx].v ;
			float m = p[idx].m ;
			/*float3*/ /*f = p[idx].f ;*/
			
			float3 down = make_float3(0.0,0.0,-1.0);
			float3 f = make_float3(0,0,0) ;
			float3 SF = make_float3(0,0,0) ;
			//p[idx].f = cud::make_float3(0,0,0) ;
			int sId;
			//printf("   spring ids (FROM)  \n : %i ,\n" , PCSpringIds[baseAddress]  ) ;
				for( int i = baseAddress ; i < baseAddress + fromCnt ; i++ )
				{
					//printf(" per particle spring ids for vert ID %i \n : %i \n" , idx ,PCSpringIds[i]  ) ;
					sId = PCSpringIds[i] ;
					SF = s[sId].springForce ;
					//if( !p[idx].fixed )
					//{
						f.x -= SF.x ;
						f.y -= SF.y ;
						f.z -= SF.z ;
					//}
					//	printf(" pts : %1.2f ,%1.2f ,%1.2f , %1.2f \n" , s[sId].springForce.x  ,s[sId].springForce.y ,s[sId].springForce.z  ) ;
				}
		
				for( int i = baseAddress + fromCnt ; i < baseAddress + fromCnt + toCnt ; i++ )
				{
					/*printf(" per particle spring ids for vert ID %i \n : %i \n" , idx ,PCSpringIds[i]  ) ;*/
					sId = PCSpringIds[i] ;
					SF = s[sId].springForce ;
					//{
						f.x += SF.x ;
						f.y += SF.y ;
						f.z += SF.z ;
					//}
					//printf(" pts : %1.2f ,%1.2f ,%1.2f , %1.2f \n" , s[sId].springForce.x  ,s[sId].springForce.y ,s[sId].springForce.z  ) ;
				}

			/*p[idx].f.x += f.x ;
			p[idx].f.y += f.y ;
			p[idx].f.z += f.z ;*/

				/* Gravitation */
			f.x += phys.gravitational * m * down.x;
			f.y += phys.gravitational * m * down.y;
			f.z += phys.gravitational *m * down.z;

				/* Viscous drag */
			f.x -= phys.viscousdrag * /*p[idx].*/v.x;
			f.y -= phys.viscousdrag * /*p[idx].*/v.y;
			f.z -= phys.viscousdrag */* p[idx].*/v.z;

			float3 dpdt = make_float3(0,0,0);
			float3 dvdt = make_float3(0,0,0);

			float oneOverMass = 1.0 / float(m);
			dpdt.x = v.x;
			dpdt.y = v.y;
			dpdt.z = v.z;
			dvdt.x = f.x * oneOverMass;
			dvdt.y = f.y * oneOverMass;
			dvdt.z = f.z * oneOverMass;

			//deriv[idx].dpdt = make_float3(dpdt.x,dpdt.y,dpdt.z );
			//deriv[idx].dvdt = make_float3(dvdt.x,dvdt.y,dvdt.z );

			//float3 pos = make_float3(0,0,0);
			//float3 vel = make_float3(0,0,0);
			/*float3*/ //dpdt = deriv[idx].dpdt ;
			/*float3*/// dvdt = deriv[idx].dvdt ;
	/*		pos.x += dpdt.x * dt;
			pos.y += dpdt.y * dt;
			pos.z += dpdt.z * dt;
			vel.x += dvdt.x * dt;
			vel.y += dvdt.y * dt;
			vel.z += dvdt.z * dt;*/

			//p[idx].f = f ;

			p[idx].p.x += dpdt.x * dt;
			p[idx].p.y += dpdt.y * dt;
			p[idx].p.z += dpdt.z * dt;

			p[idx].v.x += dvdt.x * dt;
			p[idx].v.y += dvdt.y * dt;
			p[idx].v.z += dvdt.z * dt;
		}


	}

}


__global__ void kernel_updateFixedPts( PARTICLE *p , int *fixed , int n_fixed , float ht   )
{
	int i = 0;
	for( i =0 ; i < n_fixed ; i++ )
	{
		int id = fixed[i] ;
		/*if( p[id].fixed )*/p[id].p.z = ht ;
	}
}

__global__ void kernel_calculateForces_springs_serial(PARTICLE *p,int np,
											   PARTICLEPHYS phys, PARTICLESPRING *s,int ns)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	int i,p1,p2;
	float3 down = make_float3(0.0,0.0,-1.0);
	float3 zero =  make_float3(0.0,0.0,0.0);
	float3 f;
	double len,dx,dy,dz;

	for( /*int*/ idx =0 ; idx < ns ; idx++ ) 
	{
		p1 = s[idx].from;
		p2 = s[idx].to;
		dx = p[p1].p.x - p[p2].p.x;
		dy = p[p1].p.y - p[p2].p.y;
		dz = p[p1].p.z - p[p2].p.z;
		len = sqrt(dx*dx + dy*dy + dz*dz);
		f.x  = s[idx].springconstant  * (len - s[idx].restlength);
		f.x += s[idx].dampingconstant * (p[p1].v.x - p[p2].v.x) * dx / len;
		f.x *= - dx / len;
		f.y  = s[idx].springconstant  * (len - s[idx].restlength);
		f.y += s[idx].dampingconstant * (p[p1].v.y - p[p2].v.y) * dy / len;
		f.y *= - dy / len;
		f.z  = s[idx].springconstant  * (len - s[idx].restlength);
		f.z += s[idx].dampingconstant * (p[p1].v.z - p[p2].v.z) * dz / len;
		f.z *= - dz / len;

		s[idx].springForce = make_float3(f.x,f.y,f.z);

		//if (!p[p1].fixed) 
		//{
		//	p[p1].f.x += f.x;
		//	p[p1].f.y += f.y;
		//	p[p1].f.z += f.z;
		//}
		//if (!p[p2].fixed)
		//{
		//	p[p2].f.x -= f.x;
		//	p[p2].f.y -= f.y;
		//	p[p2].f.z -= f.z;
		//}
	}
}


__global__ void kernel_calculateForces_global(PARTICLE *p,int np,
											  PARTICLEPHYS phys, PARTICLESPRING *s,int ns)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	
	//double len,dx,dy,dz;
	

	if( idx < np )
	{
		int i ;//,p1,p2;
		float3 down = make_float3(0.0,0.0,-1.0);
		//float3 zero =  make_float3(0.0,0.0,0.0);
		float3 f =  make_float3(0.0,0.0,0.0);

		p[idx].f = make_float3(0.0,0.0,0.0);
		if ( !p[idx].fixed )
		{

			float m = p[idx].m ;
			float3 v = p[idx].v ;
			/* Gravitation */
			/*p[idx].*/f.x += phys.gravitational * /*p[idx].*/m * down.x;
			/*p[idx].*/f.y += phys.gravitational * /*p[idx].*/m * down.y;
			/*p[idx].*/f.z += phys.gravitational * /*p[idx].*/m * down.z;

			/* Viscous drag */
			/*p[idx].*/f.x -= phys.viscousdrag * /*p[idx].*/v.x;
			/*p[idx].*/f.y -= phys.viscousdrag * /*p[idx].*/v.y;
			/*p[idx].*/f.z -= phys.viscousdrag */* p[idx].*/v.z;

			p[idx].f = make_float3(f.x,f.y,f.z);
			//printf(" GLOBAL : gravity %1.2f \n" , phys.gravitational);
			//printf(" GLOBAL : viscousdrag %1.2f \n" , phys.viscousdrag);
			//printf(" FORCES_GLOBAL : %1.2f ,%1.2f ,%1.2f , %1.2f \n" , p[idx].f.x ,p[idx].f.y ,p[idx].f.z, 2.0 ) ;

		}
	}
}



__global__ void kernel_calculateDerivatives(
	PARTICLE *p,int np,
	PARTICLEDERIVATIVES *deriv)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if( idx < np )
	{
		float3 dpdt = make_float3(0,0,0);
		float3 dvdt = make_float3(0,0,0);
		float3 v = p[idx].v ;
		float3 f = p[idx].f ;
		float m = p[idx].m ;

		dpdt.x = v.x;
		dpdt.y = v.y;
		dpdt.z = v.z;
		dvdt.x = f.x / m;
		dvdt.y = f.y / m;
		dvdt.z = f.z / m;

		deriv[idx].dpdt = make_float3(dpdt.x,dpdt.y,dpdt.z );
		deriv[idx].dvdt = make_float3(dvdt.x,dvdt.y,dvdt.z );
	}
}


__global__ void kernel_updatePositionAndVelocity(
					   PARTICLE *p,int np,
					   PARTICLEDERIVATIVES *deriv,double dt)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if( idx < np )
	{
		float3 pos = make_float3(0,0,0);
		float3 vel = make_float3(0,0,0);
		float3 dpdt = deriv[idx].dpdt ;
		float3 dvdt = deriv[idx].dvdt ;
		pos.x += dpdt.x * dt;
		pos.y += dpdt.y * dt;
		pos.z += dpdt.z * dt;
		vel.x += dvdt.x * dt;
		vel.y += dvdt.y * dt;
		vel.z += dvdt.z * dt;

		
		p[idx].p.x += pos.x ;
		p[idx].p.y += pos.y ;
		p[idx].p.z += pos.z ;

		p[idx].v.x += vel.x ;
		p[idx].v.y += vel.y ;
		p[idx].v.z += vel.z ;
		//printf(" pts : %1.2f ,%1.2f ,%1.2f , %1.2f \n" , p[idx].p.x ,p[idx].p.y ,p[idx].p.z, 2.0 ) ;
	}
}

__global__ void kernel_initialiseParticles(PARTICLE *p,int np)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if( idx < np )
	{
		p[idx].m = 1;
		p[idx].fixed = false;
		p[idx].v.x = 0;
		p[idx].v.y = 0;
		p[idx].v.z = 0;
	}
}

__global__ void kernel_initiliaseSprings(PARTICLE *p,int np,
										 PARTICLEPHYS phys, PARTICLESPRING *s,int ns)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if( idx < ns )
	{
		s[idx].springconstant = 1;
		s[idx].dampingconstant = 40;
		s[idx].restlength = 0.001 ;
	}

}


__global__ void kernel_fillVbo(PARTICLE *p,int np,float4 *dev_pos )
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if( idx < np )
	{
		dev_pos[idx] = make_float4(p[idx].p.x,p[idx].p.y,p[idx].p.z,p[idx].p.z); ;
	}

}

__global__ void kernel_fillVbo(PARTICLE *p,int np,float4 *dev_pos ,float4 *dev_col , bbox b0,bbox b1, bbox b2 ,bbox b3 , 
							   float3 lamp0, float3 lamp1, float3 lamp2, float3 lamp3,float angle , bool translate , bool rotate , bool scale )
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if( idx < np /*&& !p[idx].fixed*/ )
	{
		int i = 0;
		float d = 0;
		if( isInside(p[idx].p, b0 ))
		{
			point t = b0.min ; 
			i = 0 ;
			d = sqDist(p[idx].p,b0);
			transform ;
			dev_col[idx] =  make_float4(lamp0.x,lamp0.y,lamp0.z,1.0);
		}
		else if(  isInside(p[idx].p, b1 ) )
		{
			point t = b1.min ;
			i = 1 ;
			d = sqDist(p[idx].p,b1);
			transform;
			dev_col[idx] =  make_float4(lamp1.x,lamp1.y,lamp1.z,1.0);
		}
		else if(  isInside(p[idx].p, b2 ) )
		{
			point t = b2.min ;
			i = 2 ;
			d = sqDist(p[idx].p,b2);
			transform ;
			dev_col[idx] =  make_float4(lamp2.x,lamp2.y,lamp2.z,1.0);
		}
		else if(  isInside(p[idx].p, b3 ) )
		{
			point t = b3.min ;
			i = 3 ;
			d = sqDist(p[idx].p,b3);
			transform ;
			dev_col[idx] =  make_float4(lamp3.x,lamp3.y,lamp3.z,1.0);
			
		}
		else
		{

			dev_pos[idx] = make_float4(0,0,0,1);
			dev_col[idx] = make_float4(1,1,1,1);
			
		}

		if(scale)
		{
			dev_pos[idx].x *= 10 ;
			dev_pos[idx].y *= 10 ;
			dev_pos[idx].z *= 10 ;
		}
		
		
			
		
	}

}



/*
   Perform one step of the solver
*/
//#define _VERBOSE_


extern "C" void UpdateParticles(
								PARTICLE *p,int np,
								PARTICLEPHYS phys,
								PARTICLESPRING *s,int ns,
								double dt , /*PARTICLE *h_particles ,*/ float &gpuTime
								)
{
	hipEvent_t start, stop; float time , cumTime ; time = cumTime = 0;; 

	dim3 dimBlock( int(sqrt(float(nparticles))) , 1 , 1 ); 
	int thr = float(nparticles) /( dimBlock.x * dimBlock.y * dimBlock.z )  + 1 ;//9;
	dim3 dimGrid(  thr * 1 , 1 ); 


	int threads = dimBlock.x * dimBlock.y * dimBlock.y * dimGrid.x * dimGrid.y * dimGrid.z ;
	//	printf("                 total number of threads : %i \ntotal number of particles : %i \n" , threads , nparticles );
	//	printf("                 ratio - threads to particles : %i , \n" , threads / nparticles );// there should be more threads than agents, else all the agents wont get updated.
	//	printf("                 threads per block : %i , \n" , dimGrid.x );
	//	printf("                 # blocks : %i , \n" , dimBlock.x );

	//timerStart ;

		kernel_calculateForces_springs<<<  int(sqrt(float(nsprings))), int(sqrt(float(nsprings)))+2 >>>(p,np,phys,s,ns);
			checkCUDAError(" kernel_calculateForces_springs ") ;

	//timerEnd;
	//cumTime += time ;
	//printf(" GPU : kernel_calculateForces_springs : time in Milliseconds : %1.4f , cumulative Time : %1.4f \n" , time , cumTime );

	//timerStart ;

		kernel_gatherPerParticleForceAndUpdate<<<dimGrid,dimBlock>>>( p, np,s,phys, ParticleCentricSpringIds , ParticleCentricSpringIds_baseAddress , dt) ;
			checkCUDAError(" kernel_gatherPerParticleForceAndUpdate ") ;

	//cumTime += time ;
	//printf(" GPU : kernel_gatherPerParticleForceAndUpdate : time in Milliseconds : %1.4f , cumulative Time : %1.4f \n" , time , cumTime );


}





//extern "C" void UpdateParticles(
//								PARTICLE *p,int np,
//								PARTICLEPHYS phys,
//								PARTICLESPRING *s,int ns,
//								double dt , /*PARTICLE *h_particles ,*/ float &gpuTime
//								)
//{
//	hipEvent_t start, stop; float time , cumTime ; time = cumTime = 0;; 
//	
//	dim3 dimBlock( int(sqrt(float(nparticles))) , 1 , 1 ); 
//	int thr = float(nparticles) /( dimBlock.x * dimBlock.y * dimBlock.z )  + 1 ;//9;
//	dim3 dimGrid(  thr * 1 , 1 ); 
//
//
//	int threads = dimBlock.x * dimBlock.y * dimBlock.y * dimGrid.x * dimGrid.y * dimGrid.z ;
////	printf("                 total number of threads : %i \ntotal number of particles : %i \n" , threads , nparticles );
////	printf("                 ratio - threads to particles : %i , \n" , threads / nparticles );// there should be more threads than agents, else all the agents wont get updated.
////	printf("                 threads per block : %i , \n" , dimGrid.x );
////	printf("                 # blocks : %i , \n" , dimBlock.x );
//
//	
//	/*PARTICLEDERIVATIVES *deriv;*/
//
//	/*deriv = (PARTICLEDERIVATIVES *)*/
//	//hipMalloc((void**)&deriv ,np * sizeof(PARTICLEDERIVATIVES));
//	//checkCUDAError(" malloc deriv ") ;
//
//    /* Euler */
//	timerStart;
//		kernel_calculateForces_global<<<dimGrid,dimBlock>>>(p,np,phys,s,ns);
//			checkCUDAError(" kernel_calculateForces_global ") ;
//	timerEnd;
//	cumTime+=time;
//	printf(" GPU : kernel_calculateForces_global : time in Milliseconds : %1.4f \n" , time );
//
//	timerStart;
//		//kernel_calculateForces_springs_serial<<<1,1>>>(p,np,phys,s,ns);
//		kernel_calculateForces_springs<<<  int(sqrt(float(nsprings))), int(sqrt(float(nsprings)))+2 >>>(p,np,phys,s,ns);
//			checkCUDAError(" kernel_calculateForces_springs ") ;
//	timerEnd;
//	cumTime+=time;
//	printf(" GPU : kernel_calculateForces_springs : time in Milliseconds : %1.4f \n" , time );
//
//	timerStart;
//		kernel_gatherPerParticleForceAndUpdate<<<dimGrid,dimBlock>>>( p, np,s,/*deriv,*/ ParticleCentricSpringIds , ParticleCentricSpringIds_baseAddress , dt) ;
//			checkCUDAError(" kernel_gatherPerParticleForceAndUpdate ") ;
//	timerEnd;
//	cumTime+=time;
//	printf(" GPU : kernel_gatherPerParticleForceAndUpdate : time in Milliseconds : %1.4f \n" , time );
//
//	timerStart;
//		kernel_calculateDerivatives<<<dimGrid,dimBlock>>>(p,np,deriv);
//			checkCUDAError(" kernel_calculateDerivatives ") ;
//	timerEnd;
//	cumTime+=time;
//	printf(" GPU : kernel_calculateDerivatives : time in Milliseconds : %1.4f \n" , time );
//
//	timerStart;
//		kernel_updatePositionAndVelocity<<<dimGrid,dimBlock>>>( p,np,deriv,dt );
//			checkCUDAError(" kernel_updatePositionAndVelocity ") ;
//	timerEnd;
//	cumTime+=time;
//	printf( " GPU : kernel_updatePositionAndVelocity : time in Milliseconds : %1.4f \n" , time );
//
//	gpuTime = cumTime ;
//
//	//if (h_particles != NULL)
//	//	free(h_particles);
//	//h_particles = ( PARTICLE *)malloc(nparticles*sizeof(PARTICLE));
//	//printf(" GPU : UPDATE_SYSTEM - without memcpy D-H : time in Milliseconds : %1.4f \n" , cumTime );
//
//	//timerStart;
//	//hipMemcpy(h_particles,particles,nparticles * sizeof(PARTICLE),hipMemcpyDeviceToHost);
//	//	checkCUDAError( " hipMemcpy D_H ");
//	//timerEnd;
//	//cumTime+=time;
//	//printf(" GPU : CUM_TIME : time in Milliseconds : %1.4f \n" , cumTime );
//	gpuTime = cumTime ;
//
//	hipEventDestroy( start ); 
//	hipEventDestroy( stop );
//
//	//hipFree(deriv);
//
//	//kernel_calculateForces_springs<<<  int(sqrt(float(nsprings))), int(sqrt(float(nsprings)))+2 >>>(p,np,phys,s,ns);
//	//checkCUDAError(" kernel_calculateForces_springs ") ;
//
//	//kernel_gatherPerParticleForceAndUpdate<<<dimGrid,dimBlock>>>( p, np,s,/*deriv,*/ ParticleCentricSpringIds , ParticleCentricSpringIds_baseAddress , dt) ;
//	//checkCUDAError(" kernel_gatherPerParticleForceAndUpdate ") ;
//
//}
//
//



extern "C" int SetupParticles(int np,int ns)
{
   int i;

   nparticles = np;
   nsprings = ns;


   

   if (particles != NULL)
      hipFree(particles);
	hipMalloc( /*(void **)*/ &particles , nparticles * sizeof(PARTICLE) );
								checkCUDAError(" hipMalloc - particles ");
   if (springs != NULL)
      hipFree(springs);
	hipMalloc( /*(void **)*/ &springs ,nsprings * sizeof(PARTICLESPRING));
									checkCUDAError(" hipMalloc - springs ");

   physical.gravitational = 0;
   physical.viscousdrag = 0.1;

   return 1; 
}



extern "C" void InitialiseSystem( PARTICLE *h_p , int h_np , PARTICLESPRING *h_s , int h_ns , int *h_ParticleCentricSpringIds , int4 *h_ParticleCentricSpringIds_baseAddress , int h_baseAddress /* cnt*/) 
{
   int i;

	numBlocks = int(h_np / 1024) ;

	//  hipError_t result ;
	//  int device ;
	//  size_t uCurAvailMemoryInBytes;
	//  size_t uTotalMemoryInBytes;
	//  int nNoOfGPUs;
	//  hipCtx_t context;

	//  {
	//   int nID =  cu_gpuGetMaxGflopsDeviceId();
	//   hipGetDevice( &nID ); // Get handle for device
	//   hipCtxCreate( &context, 0, device ); // Create context
	//   result = hipMemGetInfo( &uCurAvailMemoryInBytes, &uTotalMemoryInBytes );

	//   if( result == hipSuccess )
	//   {
	//	   printf( "Device: %d\nTotal Memory: %d MB, Free Memory: %d MB\n",
	//		   nID,
	//		   uTotalMemoryInBytes / ( 1024 * 1024 ),
	//		   uCurAvailMemoryInBytes / ( 1024 * 1024 ));
	//   }
	//   cuDetach( context ); // Destroy context

	//  }

	//  float memNeeded =  float(h_np) * sizeof(PARTICLE) + float(h_ns) * sizeof(PARTICLESPRING) ;
	//  printf( "Device: Memory need  %1.2f MB\n",  memNeeded / ( 1024 * 1024 ));

	//if( memNeeded >= uCurAvailMemoryInBytes ) 
	//{
	//	 printf( " required memory not available, reconfigure arguments ");
	//	// return 0 ;
	//}

	printf( " -------------------------- # NP %i , # NS %i :  \n" , h_np , h_ns );
	SetupParticles(h_np,h_ns);
  
	hipMemcpy(particles,h_p,h_np*sizeof(PARTICLE),hipMemcpyHostToDevice);
		checkCUDAError("hipMemcpy particles H-D");

	hipMemcpy(springs,h_s,h_ns*sizeof(PARTICLESPRING),hipMemcpyHostToDevice);
		checkCUDAError("hipMemcpy springs H-D");
		
	hipMalloc((void**)&deriv ,h_np * sizeof(PARTICLEDERIVATIVES));
		checkCUDAError(" malloc deriv ") ;


		hipMalloc( /*(void **)*/ &particles_buf , h_np * sizeof(PARTICLE) );
		checkCUDAError(" hipMalloc - verts ");

		hipMemcpy(particles_buf,particles,h_np*sizeof(PARTICLE),hipMemcpyDeviceToDevice);
		checkCUDAError(" hipMemcpy verts H-D ");

	// ----------- 
		baseAddress = h_baseAddress ;
	if (ParticleCentricSpringIds != NULL)
		hipFree(ParticleCentricSpringIds);
	if (ParticleCentricSpringIds_baseAddress != NULL)
		hipFree(ParticleCentricSpringIds_baseAddress);


	int maxEdgesPerVertex = 8; 
	hipMalloc( &ParticleCentricSpringIds , baseAddress * sizeof( int ) );
			checkCUDAError("hipMalloc ParticleCentricSpringIds ");
	hipMalloc( &ParticleCentricSpringIds_baseAddress , h_np * sizeof( int4) );
			checkCUDAError("hipMalloc ParticleCentricSpringIds_baseAddress ");

	hipMemcpy( ParticleCentricSpringIds, h_ParticleCentricSpringIds , baseAddress * sizeof( int ) ,hipMemcpyHostToDevice ) ;
			checkCUDAError("hipMemcpy ParticleCentricSpringIds_baseAddress H-D ");
	hipMemcpy( ParticleCentricSpringIds_baseAddress, h_ParticleCentricSpringIds_baseAddress , h_np * sizeof( int4) ,hipMemcpyHostToDevice ) ;
			checkCUDAError("hipMemcpy ParticleCentricSpringIds_baseAddress H-D ");
			
	
}

extern "C" void UpdateSystem( int iterations/*PARTICLE *h_particles ,int &h_nparticles , float &gpuTime*/ /*, float3* dev_pos*/  ) 
{

	double dt = 0.01 ;
	hipEvent_t start, stop; float time , cumTime = 0;; 
	//int iterations = 10 ;

	#ifdef _VERBOSE

		printf(" -------------------------------- GPU UPDATE_NEW -------------------------------- \n ") ;
		printf( " -------------------------- # NParticles %i , # NSprings %i , #iterations %i  \n" , nparticles , nsprings ,  iterations  );
		timerStart ;
	
	#endif // _VERBOSE
	
	for( int i =0 ; i < iterations ; i++ )
		UpdateParticles(particles,nparticles,physical,springs,nsprings,dt, /*h_particles ,*/ cumTime );
	
	#ifdef _VERBOSE
		timerEnd;
		printf(" GPU : UpdateSystem : time in Milliseconds : %1.4f \n" , time );
	#endif // _VERBOSE

	//gpuTime = cumTime ;


	//printf(" -------------------------------- GPU UPDATE -------------------------------- \n ") ;
}

extern "C" void runStreams( float dt/*PARTICLE *h_particles ,int &h_nparticles , float &gpuTime*/ /*, float3* dev_pos*/  ) 
{

	/*double dt = 0.1 ;*/
	hipEvent_t start, stop; float time , cumTime = 0;; 
	int iterations = 1 ;

	#ifdef _VERBOSE

		printf(" -------------------------------- GPU UPDATE_NEW -------------------------------- \n ") ;
		printf( " -------------------------- # NParticles %i , # NSprings %i , #iterations %i  \n" , nparticles , nsprings ,  iterations  );
		timerStart ;

	#endif
	//for( int i =0 ; i < iterations ; i++ )
	//	UpdateParticles(particles,nparticles,physical,springs,nsprings,dt, /*h_particles ,*/ cumTime );


	{
		
		//kernel_clearNeighborTags<<< int( sqrt(float(nparticles))) , int( sqrt(float(nparticles)))+2 >>>( particles , nparticles  ); 
			//checkCUDAError(" kernel_clearNeighborTags ");
		kernel_moveToNearest<<< int( sqrt(float(nparticles))) , int( sqrt(float(nparticles)))+2 >>>( particles , particles_buf ,nparticles , dt ); 
			checkCUDAError(" kernel_moveToNearest ");

		hipMemcpy(particles,particles_buf,nparticles*sizeof(PARTICLE),hipMemcpyDeviceToDevice);
			checkCUDAError(" hipMemcpy verts H-D ");
	}


	#ifdef _VERBOSE

		timerEnd;
		printf(" GPU : UpdateSystem : time in Milliseconds : %1.4f \n" , time );

	#endif

	//gpuTime = cumTime ;


	//printf(" -------------------------------- GPU UPDATE -------------------------------- \n ") ;
}


extern "C" void fillVbo( float4 *dev_pos ) 
{
	kernel_fillVbo<<< int(sqrt(float(nparticles))) , int(sqrt(float(nparticles)))+2 >>>(particles,nparticles,dev_pos);
		checkCUDAError("kernel_fillVbo");
}

extern "C" void fillVbo_culled( float4 *dev_pos ,float4 *dev_col, bbox b1 , bbox b2 , bbox b3 , bbox b4 , float angle , bool translate , bool rotate , bool scale ) 
{
	kernel_fillVbo<<< int(sqrt(float(nparticles))) , int(sqrt(float(nparticles)))+2 >>>(particles,nparticles,dev_pos,dev_col, b1 ,  b2 ,  b3 ,  b4 , 
		lamp0,lamp1,lamp2,lamp3, angle , translate , rotate , scale);
	checkCUDAError("kernel_fillVbo");
}

extern "C" void updateToplogy( PARTICLE *h_p , int h_np , PARTICLESPRING *h_s , int h_ns , int *h_ParticleCentricSpringIds , int4 *h_ParticleCentricSpringIds_baseAddress , int h_baseAddress )
{
	//hipMemset(particles,0,h_np*sizeof(PARTICLE));
	//hipMemset(springs,0,h_ns*sizeof(PARTICLESPRING));

	//if( nparticles != h_np )
	//{
	//	/*if (particles != NULL)
	//		hipFree(particles);*/
	//	hipMalloc( /*(void **)*/ &particles , nparticles * sizeof(PARTICLE) );
	//	checkCUDAError(" hipMalloc - particles ");
	//}
	//if( nsprings != h_ns )
	//{
	///*	if (springs != NULL)
	//		hipFree(springs);*/
	//	hipMalloc( /*(void **)*/ &springs ,nsprings * sizeof(PARTICLESPRING));
	//	checkCUDAError(" hipMalloc - springs ");
	//}

	//if( baseAddress != h_baseAddress )
	//{
	///*	if (ParticleCentricSpringIds != NULL)
	//		hipFree(ParticleCentricSpringIds);*/
	//	hipMalloc( &ParticleCentricSpringIds , baseAddress * sizeof( int ) );
	//	checkCUDAError("hipMalloc ParticleCentricSpringIds ");
	//}

	//if( nparticles != h_np )
	//{

	//	/*if (ParticleCentricSpringIds_baseAddress != NULL)
	//		hipFree(ParticleCentricSpringIds_baseAddress);	*/
	//	hipMalloc( &ParticleCentricSpringIds_baseAddress , h_np * sizeof( int4) );
	//	checkCUDAError("hipMalloc ParticleCentricSpringIds_baseAddress ");
	//}

	nparticles = h_np ;
	nsprings= h_ns ;
	baseAddress = h_baseAddress ;

	hipMemcpy(particles,h_p,h_np*sizeof(PARTICLE),hipMemcpyHostToDevice);
		checkCUDAError("hipMemcpy particles H-D");

	hipMemcpy(springs,h_s,h_ns*sizeof(PARTICLESPRING),hipMemcpyHostToDevice);
		checkCUDAError("hipMemcpy springs H-D");

	hipMemcpy( ParticleCentricSpringIds, h_ParticleCentricSpringIds , baseAddress * sizeof( int ) ,hipMemcpyHostToDevice ) ;
		checkCUDAError("hipMemcpy ParticleCentricSpringIds_baseAddress H-D ");

	hipMemcpy( ParticleCentricSpringIds_baseAddress, h_ParticleCentricSpringIds_baseAddress , h_np * sizeof( int4) ,hipMemcpyHostToDevice ) ;
		checkCUDAError("hipMemcpy ParticleCentricSpringIds_baseAddress H-D ");

}


extern "C" void updateFixedPtsHt( float craneHts[4][1] )
{
	kernel_updateFixedPts<<<1,1>>>(particles,fixed0,n_f0,craneHts[0][1]);
		checkCUDAError("kernel_updateFixedPts");

	kernel_updateFixedPts<<<1,1>>>(particles,fixed1,n_f1,craneHts[1][1]);
		checkCUDAError("kernel_updateFixedPts");
	kernel_updateFixedPts<<<1,1>>>(particles,fixed2,n_f2,craneHts[2][1]);
		checkCUDAError("kernel_updateFixedPts");
	kernel_updateFixedPts<<<1,1>>>(particles,fixed3,n_f3,craneHts[3][1]);
		checkCUDAError("kernel_updateFixedPts");
}

extern "C" void updateLamps( float lampRgb[4][3] )
{
	lamp0 = make_float3(lampRgb[0][0],lampRgb[0][1],lampRgb[0][2]) ;
	lamp1 = make_float3(lampRgb[1][0],lampRgb[1][1],lampRgb[1][2]) ;
	lamp2 = make_float3(lampRgb[2][0],lampRgb[2][1],lampRgb[2][2]) ;
	lamp3 = make_float3(lampRgb[3][0],lampRgb[3][1],lampRgb[3][2]) ;
}
extern "C" void copyFixedPts( int *h_fixed0 ,  int *h_fixed1 ,  int *h_fixed2 , int *h_fixed3 , int h_n_f0,int h_n_f1,int h_n_f2 ,int h_n_f3 )
{

	n_f0 = h_n_f0;
	n_f1 = h_n_f1;
	n_f2 = h_n_f2;
	n_f3 = h_n_f3;
	
	{
		hipMalloc((void**)&fixed0 ,n_f0 * sizeof(int));
		checkCUDAError(" malloc fixed0 ") ;



		hipMemcpy(fixed0,h_fixed0,n_f0*sizeof(int),hipMemcpyHostToDevice);

		checkCUDAError("hipMemcpy particles H-D");
	}

	{
		hipMalloc((void**)&fixed1 ,n_f1 * sizeof(int));
		checkCUDAError(" malloc fixed0 ") ;



		hipMemcpy(fixed1,h_fixed1,n_f1*sizeof(int),hipMemcpyHostToDevice);

		checkCUDAError("hipMemcpy particles H-D");
	}

	{
		hipMalloc((void**)&fixed2,n_f2 * sizeof(int));
		checkCUDAError(" malloc fixed0 ") ;



		hipMemcpy(fixed2,h_fixed2,n_f2*sizeof(int),hipMemcpyHostToDevice);

		checkCUDAError("hipMemcpy particles H-D");
	}

	{
		hipMalloc((void**)&fixed3,n_f3 * sizeof(int));
		checkCUDAError(" malloc fixed0 ") ;



		hipMemcpy(fixed3,h_fixed3,n_f3*sizeof(int),hipMemcpyHostToDevice);

		checkCUDAError("hipMemcpy particles H-D");
	}


}

extern "C" void copyParticlePositions( PARTICLE *h_p )
{
	hipMemcpy(h_p,particles,nparticles*sizeof(PARTICLE),hipMemcpyDeviceToHost);
		checkCUDAError( "cudamemcy - particles to Host" );
}
extern "C" void CUDA_cleanup()
{

	hipFree(particles);
		checkCUDAError("hipFree : particles ");
	hipFree(springs);
		checkCUDAError("hipFree : springs ");
	hipFree(ParticleCentricSpringIds);
		checkCUDAError("hipFree : springs ");
	hipFree(ParticleCentricSpringIds_baseAddress);
		checkCUDAError("hipFree : springs ");
}