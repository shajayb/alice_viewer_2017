#include "hip/hip_runtime.h"
#ifndef MIN
#define MIN(x,y) (((x) < (y)) ? (x) : (y))
#endif

#ifndef MAX
#define MAX(x,y) (((x) > (y)) ? (x) : (y))
#endif

#include "hip/channel_descriptor.h"
#include "hip/hip_runtime_api.h"
#include "driver_functions.h"
//#include "host_defines.h"
//#include "vector_functions.h"
//#include "hip/hip_vector_types.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

#define timerStart \
	hipEventCreate(&start); \
	hipEventCreate(&stop); \
	hipEventRecord( start, 0 ); \

#define timerEnd \
	hipEventRecord( stop, 0 ); \
	hipEventSynchronize( stop ); \
	hipEventElapsedTime( &time, start, stop ); \

typedef struct 
{
	double m;      /*8 bytes*/
	float3 p;      /*12 bytes*/
	float dummy[3];/*12 bytes*/

} CUDA_STRUCT;
#define  MAX_DEVICE_ARRAY_SIZE 75000 

//////////////////////////////////////////////////////////////////////////  -------------------------------------------  CUDA UTILS

inline int cu_ConvertSMVer2Cores(int major, int minor)
{
	// Defines for GPU Architecture types (using the SM version to determine the # of cores per SM
	typedef struct {
		int SM; // 0xMm (hexidecimal notation), M = SM Major version, and m = SM minor version
		int Cores;
	} sSMtoCores;

	sSMtoCores nGpuArchCoresPerSM[] = 
	{ { 0x10,  8 }, // Tesla Generation (SM 1.0) G80 class
	  { 0x11,  8 }, // Tesla Generation (SM 1.1) G8x class
	  { 0x12,  8 }, // Tesla Generation (SM 1.2) G9x class
	  { 0x13,  8 }, // Tesla Generation (SM 1.3) GT200 class
	  { 0x20, 32 }, // Fermi Generation (SM 2.0) GF100 class
	  { 0x21, 48 }, // Fermi Generation (SM 2.1) GF10x class
	  {   -1, -1 }
	};

	int index = 0;
	while (nGpuArchCoresPerSM[index].SM != -1) {
		if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor) ) {
			return nGpuArchCoresPerSM[index].Cores;
		}	
		index++;
	}
	printf("MapSMtoCores undefined SM %d.%d is undefined (please update to the latest SDK)!\n", major, minor);
	return -1;
}
int cu_gpuGetMaxGflopsDeviceId()
{
	int current_device     = 0, sm_per_multiproc  = 0;
	int max_compute_perf   = 0, max_perf_device   = 0;
	int device_count       = 0, best_SM_arch      = 0;
	hipDeviceProp_t deviceProp;
	hipGetDeviceCount( &device_count );

	// Find the best major SM Architecture GPU device
	while (current_device < device_count)
	{
		hipGetDeviceProperties( &deviceProp, current_device );
		if (deviceProp.major > 0 && deviceProp.major < 9999)
		{
			best_SM_arch = MAX(best_SM_arch, deviceProp.major);
		}
		current_device++;
	}

	// Find the best CUDA capable GPU device
	current_device = 0;
	while( current_device < device_count )
	{
		hipGetDeviceProperties( &deviceProp, current_device );
		if (deviceProp.major == 9999 && deviceProp.minor == 9999)
		{
			sm_per_multiproc = 1;
		}
		else
		{
			sm_per_multiproc = cu_ConvertSMVer2Cores(deviceProp.major, deviceProp.minor);
		}

		int compute_perf  = deviceProp.multiProcessorCount * sm_per_multiproc * deviceProp.clockRate;

		if( compute_perf  > max_compute_perf )
		{
			// If we find GPU with SM major > 2, search only these
			if ( best_SM_arch > 2 )
			{
				// If our device==dest_SM_arch, choose this, or else pass
				if (deviceProp.major == best_SM_arch)
				{
					max_compute_perf  = compute_perf;
					max_perf_device   = current_device;
				}
			}
			else
			{
				max_compute_perf  = compute_perf;
				max_perf_device   = current_device;
			}
		}
		++current_device;
	}
	return max_perf_device;
}
static void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if( hipSuccess != err) {
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
		//exit(EXIT_FAILURE); 
	}
} 


//////////////////////////////////////////////////////////////////////////  -------------------------------------------  DEVICE VARIABLES;

CUDA_STRUCT *d_dataArray ;
CUDA_STRUCT *d_dataArray_buf ;
int d_nElems = 0; ;

//////////////////////////////////////////////////////////////////////////  -------------------------------------------  DEVICE FUNCTIONS

__device__ float Clamp(float value, float min, float max) 
{
	return value < min ? min : value > max ? max : value;
}

__device__ float Map(float value, float inputMin, float inputMax, float outputMin, float outputMax) 
{
	return ((value - inputMin) / (inputMax - inputMin) * (outputMax - outputMin) + outputMin);
}

__device__ float dot(float3 a, float3 b) 
{
	return ( (a.x * b.x) + (a.y * b.y) + (a.z * b.z) );
}
__device__ float3 normalise(float3 a) 
{
	float inv_mag = rsqrt(dot(a,a));
	return make_float3( (a.x * inv_mag) , (a.y * inv_mag) , (a.z * inv_mag) );
}





//////////////////////////////////////////////////////////////////////////  ------------------------------------------- KERNELS

__global__ void kernel_updateData(CUDA_STRUCT *data,int np )
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if( idx < np )
	{
		//d_dataArray[idx].p.x+= 0.1 ;
		//d_dataArray[idx].p.y+= 0.1 ;
		data[idx].p.z -= 0.1 ;
	}
}


__global__ void kernel_moveToNearest( CUDA_STRUCT *verts , CUDA_STRUCT *verts_buf , int nv , float disp )
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;


	if( idx < nv )
	{
		float3 p1,p2 , nearestP ;
		int nearestId ;
		float minD2 ;
		minD2 = pow(10.0f,10.0f) ;

		p1 = verts[idx].p ;

		//if( !verts[idx].fixed )
		{
			for( int j =0 ; j < nv ; j++ )
			{
				if( idx == j )continue ;

				p2 = verts[j].p ;


				float3 v = make_float3( p2.x - p1.x ,p2.y - p1.y ,p2.z - p1.z ) ;
				float d2 = ( v.x*v.x + v.y*v.y + v.z*v.z) ;/*+ 1e-16*/; // calc distance squared to emitter - make sure it's not 0

				//if( d2 > disp * 10.0 )continue;

				if( d2 < minD2 && (p2.z < p1.z) && int(verts_buf[j].dummy[0]) != (idx) )
				{
					minD2 = d2 ; 
					nearestP = p2 ;
					nearestId = j ;
				}

			}

			float3 v = make_float3( nearestP.x - p1.x ,nearestP.y - p1.y ,nearestP.z - p1.z ) ;
			float rsq = rsqrt(dot(v,v));   
			v.x *= rsq ;
			v.y *= rsq;
			v.z *= rsq ;// normalize v
			verts_buf[idx].p =  make_float3( p1.x + disp * v.x ,p1.y + disp * v.y ,p1.z + disp * v.z ) ;
			verts_buf[idx].dummy[0] = float( nearestId ); ;
		}

		//rand[idx] = verts[idx].p.z ;
		//printf(" pts : %1.2f ,%1.2f ,%1.2f  \n" , verts_buf[idx].p.x ,verts_buf[idx].p.y ,verts_buf[idx].p.z ) ;
		//rand[idx] = verts[idx].v_acc ;

	}
}
__global__ void kernel_fillVbo(CUDA_STRUCT *data,int np,float4 *devicePtr_vbo )
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if( idx < np )
	{
		devicePtr_vbo[idx] = make_float4(data[idx].p.x,data[idx].p.y,data[idx].p.z,1.0); ;
	}

}






//////////////////////////////////////////////////////////////////////////  -------------------------------------------  EXTERNS

//DATA-TRANSFER
extern "C" void initialiseDeviceMemory()
{
	if (d_dataArray != NULL)
		hipFree(d_dataArray);

	hipMalloc( /*(void **)*/ &d_dataArray , MAX_DEVICE_ARRAY_SIZE * sizeof(CUDA_STRUCT) );
	checkCUDAError(" hipMalloc - device data storage ");
	

	if (d_dataArray_buf != NULL)
		hipFree(d_dataArray_buf);

	hipMalloc( /*(void **)*/ &d_dataArray_buf , MAX_DEVICE_ARRAY_SIZE * sizeof(CUDA_STRUCT) );
	checkCUDAError(" hipMalloc - device data storage ");

	printf(" memory allocated on device : %1.2f MBytes \n" , 2.0 * (float(MAX_DEVICE_ARRAY_SIZE * sizeof(CUDA_STRUCT))/ pow(10.0f,6.0f)) ) ;
}
extern "C" void copyData_HostToDevice( CUDA_STRUCT *h_dataArray , int h_nElems )
{
	d_nElems = h_nElems ;
	hipMemcpy(d_dataArray,h_dataArray,d_nElems*sizeof(CUDA_STRUCT),hipMemcpyHostToDevice);
	checkCUDAError("hipMemcpy particles H-D");
	printf(" pts : %i \n" , d_nElems ) ;
}

extern "C" CUDA_STRUCT * copyData_DeviceToHost( CUDA_STRUCT *h_dataArray , int &n )
{
	h_dataArray = (CUDA_STRUCT *) malloc( d_nElems * sizeof(CUDA_STRUCT));
	hipMemcpy(h_dataArray,d_dataArray,d_nElems*sizeof(CUDA_STRUCT),hipMemcpyDeviceToHost);
	checkCUDAError("hipMemcpy particles H-D");
	n = d_nElems ;
	return h_dataArray ;
}
// UPDATE DATA

extern "C" void updateDeviceData()
{
	

	hipEvent_t start, stop; 
	float time , cumTime ; time = cumTime = 0;; 

	timerStart ;

	kernel_updateData<<< int(sqrt(float(d_nElems))) , int(sqrt(float(d_nElems)))+2 >>>(d_dataArray,d_nElems);
	checkCUDAError("kernel_updateData");
	
	timerEnd;
	cumTime += time ;
	//printf(" GPU : kernel_updateData : time in ms : %1.4f , cumulative Time : %1.4f \n" , time , cumTime );

}

extern "C" float runStreams()
{
	hipEvent_t start, stop; 
	float time , cumTime ; time = cumTime = 0;; 
	
	timerStart ;

		kernel_moveToNearest<<< int( sqrt(float(d_nElems))) , int( sqrt(float(d_nElems)))+2 >>>( d_dataArray , d_dataArray_buf ,d_nElems , 0.01 ); 
		checkCUDAError(" kernel_moveToNearest ");

		hipMemcpy(d_dataArray,d_dataArray_buf,d_nElems*sizeof(CUDA_STRUCT),hipMemcpyDeviceToDevice);
		checkCUDAError(" hipMemcpy verts H-D ");

	timerEnd;
	cumTime += time ;
	//printf(" GPU : kernel_updateData : time in ms : %1.4f , cumulative Time : %1.4f \n" , time , cumTime );
	return cumTime;
}
//DISPLAY
extern "C" void fillVbo( float4 *dev_pos ) 
{
	kernel_fillVbo<<< int(sqrt(float(d_nElems))) , int(sqrt(float(d_nElems)))+2 >>>(d_dataArray,d_nElems,dev_pos);
	checkCUDAError("kernel_fillVbo");
}

//EXIT

extern "C" void CUDA_cleanup()
{
	hipFree(d_dataArray);
	checkCUDAError("hipFree : device data ");
}
